
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

#define CHECK(call){ \ 
    const hipError_t cuda_ret = call; \
    if (cuda_ret != hipSuccess){ \
        printf("Error: %s:%d, ",__FILE__, __LINE__); \
        printf("code: %d, reason %s\n", cuda_ret, hipGetErrorString(cuda_ret)); \
        exit(-1); \
    } \
}